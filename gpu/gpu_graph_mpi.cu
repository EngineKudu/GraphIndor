#include "hip/hip_runtime.h"
#include <schedule_IEP.h>
#include <mpi.h>
#include <omp.h>

#include <unistd.h>
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <atomic>
#include <tuple>

#include <utility>

struct SpinLock {
    std::atomic_flag flag;

    SpinLock() : flag{ATOMIC_FLAG_INIT} {}
    void lock() { while (flag.test_and_set()) asm volatile ("pause"); }
    void unlock() { flag.clear(); }
};

template <typename Lock>
struct LockGuard {
    Lock &_lock;
    LockGuard(Lock &lock) : _lock{lock} { _lock.lock(); }
    ~LockGuard() { _lock.unlock(); }
};

__global__ void cuda_kernel(int node)
{
    printf("hello from cuda thread=%d block=%d got rank=%d\n", threadIdx.x, blockIdx.x, node);
}

__global__ void spin_kernel(clock_t cycles)
{
    clock_t start = clock64();
    while (clock64() - start < cycles)
        ;
    printf("spin_kernel done. thread=%d\n", threadIdx.x);
}

// #define log(fmt, ...) fprintf(stderr, fmt, ##__VA_ARGS__)
#define log(fmt, ...) (void)(fmt)

void do_work(int task_id)
{
    // int us = 10000 + rand() % 10000;
    int us = 100 + rand() % 100;
    usleep(us);

    int tid = omp_get_thread_num();
    log("omp thread %d finish task %d\n", tid, task_id);
}

// shared by all threads
int nr_tasks = 10000;
using TaskStatus = std::tuple<int, int>;
TaskStatus task_status;
SpinLock task_status_lock;
volatile bool workers_should_stop = false;
volatile bool task_requested = false;
bool *workers_idle;

constexpr int CPU_WORKER_TASK_GRANULARITY = 2;
constexpr int GPU_WORKER_TASK_GRANULARITY = 5;
constexpr int NODE_TASK_GRANULARUTY = 10;

enum MessageType {
    MSG_REQUEST_WORK,  // slave -> master
    MSG_DISPATCH_WORK, // master -> slave
    MSG_REPORT_ANS     // slave -> master
};

enum NodeState {
    WORKING, // normal working state
    WAITING  // no more work from master node, waiting for worker threads to finish
};

NodeState state = WORKING; // only used by scheduler thread
int global_cur_task; // only scheduler thread of master node will modify this var in working phase
int global_ans = 0, nr_idle_nodes = 0, gpu_ans = 0;
std::atomic<int> node_ans{0};

void process_message(uint64_t recv_buf[], uint64_t send_buf[], int node, int sender)
{
    MPI_Request send_req;
    switch (recv_buf[0]) {
    case MSG_REQUEST_WORK: { // me: master
        send_buf[0] = MSG_DISPATCH_WORK;
        send_buf[1] = global_cur_task;
        send_buf[2] = global_cur_task = std::min(global_cur_task + NODE_TASK_GRANULARUTY, nr_tasks);
        MPI_Isend(send_buf, 3, MPI_UINT64_T, sender, 0, MPI_COMM_WORLD, &send_req);
        log("master got work request from node %d, replying [%ld, %ld)\n", sender, send_buf[1], send_buf[2]);
        break;
    }
    case MSG_DISPATCH_WORK: { // me: slave
        int new_task_cur = recv_buf[1];
        int new_task_end = recv_buf[2];
        if (new_task_cur >= nr_tasks) {
            state = NodeState::WAITING;
            log("slave node %d enters WAITING state\n", node);
        } else {
            LockGuard<SpinLock> guard{task_status_lock};
            task_status = std::make_tuple(new_task_cur, new_task_end);
            log("slave node %d got task [%d, %d)\n", node, new_task_cur, new_task_end);
            task_requested = false;
        }
        break;
    }
    case MSG_REPORT_ANS: { // me: master
        ++nr_idle_nodes;
        global_ans += recv_buf[1];
        log("master receive answer %ld from node %d\n", recv_buf[1], sender);
        break;
    }
    }
}

// TODO: require lock?
bool all_workers_idle()
{
    int nr_threads = omp_get_max_threads();
    int idle_count = 0;
    for (int i = 0; i < nr_threads; ++i)
        if (workers_idle[i])
            ++idle_count;
    return idle_count == nr_threads;
}

// TODO: returns true when too many worker threads are idle?
bool should_request_work()
{
    LockGuard<SpinLock> guard{task_status_lock};
    return std::get<0>(task_status) >= std::get<1>(task_status);
}

// returns whether task status is successfully updated
bool update_task_range(std::tuple<int, int>& task_range, int max_nr_tasks)
{
    int task_cur, task_end, new_task_cur;
    LockGuard<SpinLock> guard{task_status_lock};
    std::tie(task_cur, task_end) = task_status;
    if (task_cur < task_end) {
        new_task_cur = std::min(task_cur + max_nr_tasks, task_end);
        task_range = std::make_tuple(task_cur, new_task_cur);
        task_status = std::make_tuple(new_task_cur, task_end);
        return true;
    }
    return false;
}

// thread 0 is scheduler
void scheduler_loop(int comm_sz, int node)
{
    hipEvent_t event;
    hipEventCreate(&event);
    TaskStatus gpu_task_range;
    bool gpu_working = false;

    constexpr int MSG_BUF_LEN = 256;
    static uint64_t recv_buf[MSG_BUF_LEN], send_buf[MSG_BUF_LEN];
    MPI_Request send_req, recv_req;
    MPI_Status mpi_status;

    MPI_Irecv(recv_buf, MSG_BUF_LEN, MPI_UINT64_T, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &recv_req);
    while (true) {
        if (node == 0 || state == NodeState::WORKING) {
            int msg_received = 0;
            MPI_Test(&recv_req, &msg_received, &mpi_status);
            if (msg_received) {
                process_message(recv_buf, send_buf, node, mpi_status.MPI_SOURCE);
                MPI_Irecv(recv_buf, MSG_BUF_LEN, MPI_UINT64_T, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &recv_req);
            }
        }
        
        switch (state) {
        case NodeState::WORKING: {
            if (should_request_work()) {
                if (node != 0) {
                    if (!task_requested) {
                        send_buf[0] = MSG_REQUEST_WORK;
                        MPI_Isend(send_buf, 1, MPI_UINT64_T, 0, 0, MPI_COMM_WORLD, &send_req);
                        task_requested = true;
                    }
                } else {
                    int new_task_cur, new_task_end;
                    new_task_cur = global_cur_task;
                    new_task_end = global_cur_task = std::min(global_cur_task + NODE_TASK_GRANULARUTY, nr_tasks);
                    if (new_task_cur >= nr_tasks) {
                        state = NodeState::WAITING;
                        log("master node enters WAITING state\n");
                    } else {
                        LockGuard<SpinLock> guard{task_status_lock};
                        task_status = std::make_tuple(new_task_cur, new_task_end);
                        log("master node got task [%d, %d)\n", new_task_cur, new_task_end);
                    }
                }
            }
            break;
        }
        case NodeState::WAITING: {
            if (all_workers_idle()) {
                workers_should_stop = true;
                if (node != 0) {
                    send_buf[0] = MSG_REPORT_ANS;
                    send_buf[1] = node_ans;
                    MPI_Isend(send_buf, 2, MPI_UINT64_T, 0, 0, MPI_COMM_WORLD, &send_req);
                    return;
                } else {
                    if (nr_idle_nodes == comm_sz - 1)
                        return;
                }
            }
            break;
        }
        }

        if (!gpu_working) {
            if (update_task_range(gpu_task_range, GPU_WORKER_TASK_GRANULARITY)) {
                // launch cuda kernel
                spin_kernel<<<1, 4>>>(1000000);
                hipEventRecord(event);
                gpu_working = true;
                workers_idle[0] = false;
                log("node %d gpu kernel launched. [%d, %d)\n", node, std::get<0>(gpu_task_range), std::get<1>(gpu_task_range));
            }
        } else { // poll gpu task state
            auto result = hipEventQuery(event);
            if (hipErrorNotReady == result)
                continue;
            
            assert(hipSuccess == result);
            hipDeviceSynchronize();
            gpu_working = false;
            workers_idle[0] = true;
            node_ans += std::get<1>(gpu_task_range) - std::get<0>(gpu_task_range);
            gpu_ans  += std::get<1>(gpu_task_range) - std::get<0>(gpu_task_range);
        }
    }
}

// other threads are workers
void worker_loop(int node)
{
    int thread_id = omp_get_thread_num();
    while (!workers_should_stop) {
        TaskStatus task_range;
        bool idle = !update_task_range(task_range, CPU_WORKER_TASK_GRANULARITY);
        if (idle) {
            workers_idle[thread_id] = true;
            continue;
        }
        workers_idle[thread_id] = false;
        
        int task_begin, task_end;
        std::tie(task_begin, task_end) = task_range;
        // for (int i = task_begin; i < task_end; ++i)
        //     do_work(i);
        log("node %d thread %d do work [%d, %d)\n", node, thread_id, task_begin, task_end);
        if (node)
            usleep(100);
        else
            usleep(10000);
        node_ans += task_end - task_begin;
    }
}

void test_cuda_event()
{
    hipEvent_t event;
    hipEventCreate(&event);
    spin_kernel<<<1, 32>>>(1000000000); // ~ 1s, 1e9
    hipEventRecord(event);
    while (true) {
        auto result = hipEventQuery(event);
        if (hipSuccess == result)
            break;
        if (hipErrorNotReady == result) {
            printf("waiting for device...\n");
            usleep(100000);
        } else {
            printf("oops.. %s\n", hipGetErrorString(result));
            break;
        }
    }
}

int main(int argc, char* argv[])
{
    int comm_sz, node;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_sz);
    MPI_Comm_rank(MPI_COMM_WORLD, &node);
    fprintf(stderr, "size = %d rank = %d\n", comm_sz, node);

    int nr_threads = omp_get_max_threads();
    workers_idle = new bool[nr_threads] {true};
    // init task_status, global_cur_task
    task_status = std::make_tuple(node * 10, node * 10 + 10);
    global_cur_task = comm_sz * 10;

    #pragma omp parallel
    {
        int thread_id = omp_get_thread_num();
        if (thread_id == 0) {
            scheduler_loop(comm_sz, node);
        } else {
            worker_loop(node);
        }
        log("node %d thread %d finish.\n", node, thread_id);
    }
    
    MPI_Finalize();
    if (node == 0)
        printf("final answer = %d\n", global_ans + node_ans);
    printf("node %d node_ans = %d gpu_ans = %d\n", node, int(node_ans), gpu_ans);
    return 0;
}
