#include "hip/hip_runtime.h"
// This program can only count House pattern using GPU.
#include <graph.h>
#include <dataloader.h>
#include <vertex_set.h>
#include <common.h>

#include <assert.h>
#include <iostream>
#include <string>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <>

#include <sys/time.h>
class TimeInterval{
public:
    TimeInterval(){
        check();
    }

    void check(){
        gettimeofday(&tp, NULL);
    }

    void print(const char* title){
        struct timeval tp_end, tp_res;
        gettimeofday(&tp_end, NULL);
        timersub(&tp_end, &tp, &tp_res);
        std::cout << title << ": " << tp_res.tv_sec << " s " << tp_res.tv_usec << " us.\n";
    }
private:
    struct timeval tp;
};

TimeInterval allTime;
TimeInterval tmpTime;

#define COUNT_INTERSECTION 0
#define COUNT_TIME 1

#if COUNT_TIME
__device__ uint32_t i_count = 0;
__device__ uint64_t i_time = 0;

__device__ static inline uint64_t read_timer()
{
    // Due to a bug in CUDA's 64-bit globaltimer, the lower 32 bits can wrap
    // around after the upper bits have already been read. Work around this by
    // reading the high bits a second time. Use the second value to detect a
    // rollover, and set the lower bits of the 64-bit "timer reading" to 0, which
    // would be valid, it's passed over during the duration of the reading. If no
    // rollover occurred, just return the initial reading.
    volatile uint64_t first_reading;
    volatile uint32_t second_reading;
    uint32_t high_bits_first;
    asm volatile ("mov.u64 %0, %%globaltimer;" : "=l"(first_reading));
    high_bits_first = first_reading >> 32;
    asm volatile ("mov.u32 %0, %%globaltimer_hi;" : "=r"(second_reading));
    if (high_bits_first == second_reading) {
        return first_reading;
    }
    // Return the value with the updated high bits, but the low bits set to 0.
    return ((uint64_t) second_reading) << 32;
}
#endif

#if COUNT_INTERSECTION
__device__ uint64_t i_inter_size_0 = 0, i_inter_time_0 = 0;
__device__ uint64_t i_inter_size_1 = 0, i_inter_time_1 = 0;
__device__ uint64_t i_inter_size_2 = 0, i_inter_time_2 = 0;
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define THREADS_PER_BLOCK 32

__device__ unsigned long long dev_sum;
__device__ unsigned int dev_nowEdge;

__device__ void intersection(uint32_t *out, uint32_t *a, uint32_t *b, uint32_t na, uint32_t nb, uint32_t* result_size) {
#if COUNT_TIME
        auto  t1 = read_timer();
#endif
    __shared__ uint32_t out_offset[THREADS_PER_BLOCK];
    __shared__ uint32_t out_size;
    
    int lid = threadIdx.x;
    
    if (lid == 0)
        out_size = 0;

    uint32_t num_done = 0;
    while (num_done < na) {
        bool found = 0;
        uint32_t u = 0;
        if (num_done + lid < na) {
            int mid, l = 0, r = nb - 1; // [l, r], use signed int instead of unsigned int!
            u = a[num_done + lid]; // u: an element in set a
            while (l <= r) {
                mid = (l + r) >> 1;
                if (b[mid] < u) {
                    l = mid + 1;
                } else if (b[mid] > u) {
                    r = mid - 1;
                } else {
                    found = 1;
                    break;
                }
            }
        }
        out_offset[lid] = found;
        __syncthreads();

        for (int s = 1; s < THREADS_PER_BLOCK; s *= 2) {
            uint32_t v = lid >= s ? out_offset[lid - s] : 0;
            __syncthreads();
            out_offset[lid] += v;
            __syncthreads();
        }

        if (found) {
            uint32_t offset = out_offset[lid] - 1;
            out[out_size + offset] = u;
        }

        if (lid == 0)
            out_size += out_offset[THREADS_PER_BLOCK - 1];
        num_done += THREADS_PER_BLOCK;
    }
    __syncthreads();
    if( lid == 0)
        *result_size = out_size;
#if COUNT_TIME
    auto t3 = read_timer();
    if (threadIdx.x % 32 == 0) {
        atomicAdd((unsigned long long*)&i_time, t3 - t1);
    }
#endif
    
}

__device__ void detect_v2(uint32_t *tmp, uint32_t size, uint32_t v2, bool *p_hit) {
    uint32_t i = 0;
    bool hit = false;
    while(i + threadIdx.x < size) {
        hit |= tmp[i+threadIdx.x] == v2;
        i += THREADS_PER_BLOCK;
    }
    if(hit) *p_hit = true;
}

__device__ void upd_ans(uint32_t *a, uint32_t *b, uint32_t na, uint32_t nb, unsigned long long *p_sum) {
    __shared__ uint32_t out_size[32];
    
    int lid = threadIdx.x;
#if COUNT_INTERSECTION
    if(lid == 0) {
        atomicAdd((unsigned long long*)&i_inter_size_1, (uint64_t)na + nb);
        atomicAdd((unsigned long long*)&i_inter_time_1, (uint64_t)1);
    }
#endif
    
    out_size[lid] = 0;

    uint32_t num_done = 0;
    while (num_done < na) {
        bool found = 0;
        uint32_t u = 0;
        if (num_done + lid < na) {
            int mid, l = 0, r = nb - 1; // [l, r], use signed int instead of unsigned int!
            u = a[num_done + lid]; // u: an element in set a
            while (l <= r) {
                mid = (l + r) >> 1;
                if (b[mid] < u) {
                    l = mid + 1;
                } else if (b[mid] > u) {
                    r = mid - 1;
                } else {
                    found = 1;
                    break;
                }
            }
        }
        out_size[lid] += found;
        num_done += THREADS_PER_BLOCK;
    }
   __syncthreads();

    for (int s = 1; s < THREADS_PER_BLOCK; s *= 2) {
        uint32_t v = lid >= s ? out_size[lid - s] : 0;
        __syncthreads();
        out_size[lid] += v;
        __syncthreads();
    }
    if(lid == 0)
        *p_sum -= out_size[31];

    /*   __shared__ uint32_t lblock[THREADS_PER_BLOCK];
         __shared__ uint32_t rblock[THREADS_PER_BLOCK];

         uint32_t i = 0, j = 0;
         unsigned long long sum = 0;
         uint32_t lsize = THREADS_PER_BLOCK, rsize = THREADS_PER_BLOCK;

         while (i < ln && j < rn) {

         lsize = min(ln - i, THREADS_PER_BLOCK);
         rsize = min(rn - j, THREADS_PER_BLOCK);

         if(i + threadIdx.x < ln) lblock[threadIdx.x] = lbases[i + threadIdx.x];
         if(j + threadIdx.x < rn) rblock[threadIdx.x] = rbases[j + threadIdx.x];

         __threadfence_block();

         for(int k = 0; k < rsize; ++k)
         sum += (threadIdx.x < lsize) & (lblock[threadIdx.x] == rblock[k]);

         uint32_t llast = lblock[lsize - 1];
         uint32_t rlast = rblock[rsize - 1];

         if(llast >= rlast) j += rsize;
         if(llast <= rlast) i += lsize;
         }

         (*p_sum) -= sum;*/
    /*    i = 0;
          j = 0;
          unsigned long long size = 0;
          while(i < ln && j < rn) {
          if(lbases[i]==rbases[j] && i % 32 == threadIdx.x) {
          ++size;
          }
          int u = lbases[i],v=rbases[j];
          i+=u<=v;
          j+=v<=u;
          }
          assert(size==sum);*/

}

__global__ void __dfs(uint32_t edge_num, uint32_t buffer_size, uint32_t *edge_from, uint32_t *edge, uint32_t *vertex, uint32_t *tmp) {
    __shared__ unsigned int edgeI;
    __shared__ unsigned int edgeEnd;
    __shared__ unsigned long long sdata[THREADS_PER_BLOCK];

    unsigned long long mysum = 0;

    uint32_t tmp1_begin = buffer_size * 2 * blockIdx.x;
    uint32_t tmp2_begin = tmp1_begin + buffer_size;

    if(threadIdx.x == 0) {
        edgeI = edgeEnd = 0;
    }

    __syncthreads();

    assert( edgeI == edgeEnd);

    uint32_t v0,v1,v2;
    __shared__ uint32_t tmp1_size;
    __shared__ uint32_t tmp2_size;

    uint32_t lb,le,ln;
    uint32_t rb,re,rn;

    uint32_t loop_begin, loop_limit;

    __shared__ bool have_v2;

#if COUNT_INTERSECTION
    int lid = threadIdx.x;
#endif
#if COUNT_TIME
    bool do_work = false;
#endif

    while(true) {
        if(threadIdx.x == 0) {
            //printf("%d at %u\n", blockIdx.x, edgeI);
            if(++edgeI >= edgeEnd) {
                edgeI = atomicAdd(&dev_nowEdge, 1);
                edgeEnd = min(edge_num, edgeI + 1);
            }
        }

        __syncthreads();

        unsigned int i = edgeI;
        if(i >= edge_num) break;
#if COUNT_TIME
        do_work = true;
#endif

        // for edge in E
        v0 = edge_from[i];
        v1 = edge[i];

        if(v0 < v1) continue; // optimize

        lb = vertex[v0];
        le = vertex[v0+1];
        ln = le - lb;

        rb = vertex[v1];
        re = vertex[v1+1];
        rn = re - rb;
#if COUNT_INTERSECTION
    if(lid == 0) {
        atomicAdd((unsigned long long*)&i_inter_size_0, (uint64_t)ln + rn);
        atomicAdd((unsigned long long*)&i_inter_time_0, (uint64_t)1);
    }
#endif

        intersection(tmp + tmp1_begin, edge + lb, edge + rb, ln, rn, &tmp1_size); // v3's set = tmp1 = N(v0) & N(v1)
        __syncthreads();
        
        if(tmp1_size == 0) continue;

        loop_begin = vertex[v0];
        loop_limit = vertex[v0+1];
        for(uint32_t j = loop_begin; j < loop_limit; ++j) {
            v2 = edge[j]; // for v2 in N(v1)
            if(v1==v2) continue;

            lb = vertex[v2];
            le = vertex[v2+1];
            ln = le - lb;
#if COUNT_INTERSECTION
    if(lid == 0) {
        atomicAdd((unsigned long long*)&i_inter_size_2, (uint64_t)ln + rn);
        atomicAdd((unsigned long long*)&i_inter_time_2, (uint64_t)1);
    }
#endif

            intersection(tmp + tmp2_begin, edge + lb, edge + rb, ln, rn, &tmp2_size); // v4's set = tmp2 = N(v0) & N(v2)
            __syncthreads();

            if(tmp2_size <= 1) continue;

            if(threadIdx.x == 0) {
                have_v2 = false;
            }
            __syncthreads();

            detect_v2(tmp + tmp1_begin, tmp1_size, v2, &have_v2); // notice that v2 may belong to tmp1, but we want tmp1 - {v2}

            upd_ans(tmp + tmp1_begin, tmp + tmp2_begin, tmp1_size, tmp2_size, &mysum); // ans -= (tmp1 & tmp2).size
            __syncthreads();

            if(threadIdx.x == 0) { // ans += tmp1.size * tmp2.size, notice that v1 always exist in tmp2, so we use tmp2_size-1
                if(have_v2) mysum += (tmp1_size - 1) * (tmp2_size - 1);
                else mysum += tmp1_size * (tmp2_size - 1);
            }
        }
    }

    sdata[threadIdx.x] = mysum;
    __syncthreads();

    for (int s=1; s < blockDim.x; s *=2){
        int index = 2 * s * threadIdx.x;

        if (index < blockDim.x){
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&dev_sum, sdata[0]);
#if COUNT_TIME
        if(do_work) atomicAdd(&i_count, 1);
#endif
    }
    
}

void gpu_pattern_matching(Graph *g) {
    uint32_t *edge_from = new uint32_t[g->e_cnt];
    for(uint32_t i = 0; i < g->v_cnt; ++i)
        for(uint32_t j = g->vertex[i]; j < g->vertex[i+1]; ++j)
            edge_from[j] = i;

    uint32_t *edge = new uint32_t[g->e_cnt];
    uint32_t *vertex = new uint32_t[g->v_cnt + 1];

    for(uint32_t i = 0;i < g->e_cnt; ++i) edge[i] = g->edge[i];
    for(uint32_t i = 0;i <= g->v_cnt; ++i) vertex[i] = g->vertex[i];

    tmpTime.check(); 
    int numBlocks = 4096;

    uint32_t size_edge = g->e_cnt * sizeof(uint32_t);
    uint32_t size_vertex = (g->v_cnt + 1) * sizeof(uint32_t);
    uint32_t size_tmp  = VertexSet::max_intersection_size * sizeof(uint32_t) * numBlocks * (1 + 1);

    uint32_t *dev_edge;
    uint32_t *dev_edge_from;
    uint32_t *dev_vertex;
    uint32_t *dev_tmp;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_edge_from, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_vertex, size_vertex));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));

    gpuErrchk( hipMemcpy(dev_edge, edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_edge_from, edge_from, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_vertex, vertex, size_vertex, hipMemcpyHostToDevice));

    unsigned long long sum = 0;

    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_sum), &sum, sizeof(unsigned long long), 0, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_nowEdge), &sum, sizeof(unsigned int), 0, hipMemcpyHostToDevice));

    tmpTime.print("Prepare time cost");
    tmpTime.check();

    uint32_t edge_num = g->e_cnt;
    uint32_t buffer_size = VertexSet::max_intersection_size;
    __dfs<<<numBlocks, THREADS_PER_BLOCK>>>(edge_num, buffer_size, dev_edge_from, dev_edge, dev_vertex, dev_tmp);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

#if COUNT_INTERSECTION
    uint64_t i_inter_size_cpu_0, i_inter_time_cpu_0;
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_size_cpu_0, HIP_SYMBOL(i_inter_size_0), sizeof(i_inter_size_0)));
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_time_cpu_0, HIP_SYMBOL(i_inter_time_0), sizeof(i_inter_time_0)));
    printf("inter time0 %llu, inter size0 %llu \n", i_inter_time_cpu_0, i_inter_size_cpu_0);
    
    uint64_t i_inter_size_cpu_1, i_inter_time_cpu_1;
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_size_cpu_1, HIP_SYMBOL(i_inter_size_1), sizeof(i_inter_size_1)));
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_time_cpu_1, HIP_SYMBOL(i_inter_time_1), sizeof(i_inter_time_1)));
    printf("inter time1 %llu, inter size1 %llu \n", i_inter_time_cpu_1, i_inter_size_cpu_1);
    
    uint64_t i_inter_size_cpu_2, i_inter_time_cpu_2;
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_size_cpu_2, HIP_SYMBOL(i_inter_size_2), sizeof(i_inter_size_2)));
    gpuErrchk( hipMemcpyFromSymbol(&i_inter_time_cpu_2, HIP_SYMBOL(i_inter_time_2), sizeof(i_inter_time_2)));
    printf("inter time2 %llu, inter size2 %llu \n", i_inter_time_cpu_2, i_inter_size_cpu_2);
#endif

#if COUNT_TIME
    int i_count_cpu;
    uint64_t t_time_cpu, i_time_cpu, ihead_time_cpu;
    gpuErrchk( hipMemcpyFromSymbol(&i_count_cpu, HIP_SYMBOL(i_count), sizeof(i_count)));
    gpuErrchk( hipMemcpyFromSymbol(&i_time_cpu, HIP_SYMBOL(i_time), sizeof(i_time)));
    printf("Warp count: %d\nIntersection time: %ld\n", i_count_cpu, i_time_cpu);
#endif

    printf("house count %llu\n", sum);
    tmpTime.print("Counting time cost");
}

int main(int argc,char *argv[]) {
    Graph *g;
    DataLoader D;

    /*
    const std::string type = argv[1];
    const std::string path = argv[2];

    DataType my_type;

    GetDataType(my_type, type);

    if(my_type == DataType::Invalid) {
        printf("Dataset not found!\n");
        return 0;
    }

    D.load_data(g,my_type,path.c_str());
    //assert(D.load_data(g,my_type,path.c_str())==true); 

 //   assert(D.load_data(g,10));
    printf("Load data success!\n");
    fflush(stdout);
    */

    bool ok = D.fast_load(g, argv[1]);

    if (!ok) {
        printf("data load failure :-(\n");
        return 0;
    }

    allTime.check();

    gpu_pattern_matching(g);

    allTime.print("Total time cost");

    return 0;
}
