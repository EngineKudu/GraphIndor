#include "hip/hip_runtime.h"
__global__ void gpu_pattern_matching(uint32_t edge_num, uint32_t buffer_size, uint32_t *edge_from, uint32_t *edge, uint32_t *vertex, uint32_t *tmp, const GPUSchedule* schedule) {
    __shared__ unsigned int block_edge_idx[WARPS_PER_BLOCK];
    extern __shared__ GPUVertexSet block_vertex_set[];
    extern __shared__ char block_shmem[];

    int wid = threadIdx.x / THREADS_PER_WARP;
    int lid = threadIdx.x % THREADS_PER_WARP;
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid;
    unsigned int &edge_idx = block_edge_idx[wid];
    GPUVertexSet *vertex_set = block_vertex_set + wid * 4;

    GPUVertexSet &subtraction_set = vertex_set[3];
    if (lid == 0) {
        edge_idx = 0;
        uint32_t offset = buffer_size * global_wid * 3;

        uint32_t *block_subtraction_set_buf = (uint32_t *)(block_shmem + 512);
        subtraction_set.set_data_ptr(block_subtraction_set_buf + wid * 3);

        for (int i = 0; i < 3; ++i) {
            vertex_set[i].set_data_ptr(tmp + offset);
            offset += buffer_size;
        }
    }

    __threadfence_block();

    uint32_t v0, v1, v2;
    uint32_t l, r;
    unsigned long long sum = 0;

    while (true) {
        if (lid == 0) {
            edge_idx = atomicAdd(&dev_cur_edge, 1);
        }
        __threadfence_block();

        unsigned int i = edge_idx;
        if (i >= edge_num) break;

        v0 = edge_from[i];
        v1 = edge[i];
        if (v0 <= v1) continue;

        get_edge_index(v0, l, r);
        if (threadIdx.x % THREADS_PER_WARP == 0)
            vertex_set[0].init(r - l, &edge[l]);
        __threadfence_block();
        
        get_edge_index(v1, l, r);
        GPUVertexSet* tmp_vset;
        intersection2(vertex_set[1].get_data_ptr(), vertex_set[0].get_data_ptr(), &edge[l], vertex_set[0].get_size(), r - l, &vertex_set[1].size);
        if (vertex_set[1].get_size() == 0) continue;
        
        int loop_size_depth2 = vertex_set[1].get_size();
        uint32_t* loop_data_ptr_depth2 = vertex_set[1].get_data_ptr();
        for (int i_depth2 = 0; i_depth2 < loop_size_depth2; ++i_depth2) {
            uint32_t v_depth2 = loop_data_ptr_depth2[i_depth2];
            if (v0 == v_depth2 || v1 == v_depth2) continue;

            unsigned int l_depth2, r_depth2;
            get_edge_index(v_depth2, l_depth2, r_depth2);
            {
                tmp_vset = &vertex_set[2];
                if (threadIdx.x % THREADS_PER_WARP == 0)
                    tmp_vset->init(r_depth2 - l_depth2, &edge[l_depth2]);
                __threadfence_block();
                if (r_depth2 - l_depth2 > vertex_set[1].get_size())
                    tmp_vset->size -= unordered_subtraction_size(*tmp_vset, vertex_set[1], -1);
                else
                    tmp_vset->size = vertex_set[1].get_size() - unordered_subtraction_size(vertex_set[1], *tmp_vset, -1);
            }
            if (vertex_set[2].get_size() == 0) continue;
            
            v2 = v_depth2; // subtraction_set.push_back(v2);

            int ans0 = vertex_set[2].get_size() - 0;
            int ans1 = vertex_set[1].get_size() - 1;
            long long val;
            val = ans0;
            val = val * ans0;
            val = val * ans1;
            sum += val * 1;
            val = ans0;
            val = val * ans0;
            sum += val * -1;
            val = ans0;
            val = val * ans0;
            sum += val * -1;
            val = ans0;
            val = val * ans1;
            sum += val * -1;
            val = ans0;
            sum += val * 2;
        }
    }
    if (lid == 0) atomicAdd(&dev_sum, sum);
}
