#include "hip/hip_runtime.h"
void pattern_matching_init(Graph *g, const Schedule_IEP& schedule_iep) {
    printf("basic prefix %d, total prefix %d\n", schedule_iep.get_basic_prefix_num(), schedule_iep.get_total_prefix_num());

    int num_blocks = 1024;
    int num_total_warps = num_blocks * WARPS_PER_BLOCK;

    size_t size_edge = g->e_cnt * sizeof(uint32_t);
    size_t size_vertex = (g->v_cnt + 1) * sizeof(uint32_t);
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * (schedule_iep.get_total_prefix_num() + 2); //prefix + subtraction + tmp

    schedule_iep.print_schedule();
    uint32_t *edge_from = new uint32_t[g->e_cnt];
    for(uint32_t i = 0; i < g->v_cnt; ++i)
        for(uint32_t j = g->vertex[i]; j < g->vertex[i+1]; ++j)
            edge_from[j] = i;

    tmpTime.check(); 

    uint32_t *dev_edge;
    uint32_t *dev_edge_from;
    uint32_t *dev_vertex;
    uint32_t *dev_tmp;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_edge_from, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_vertex, size_vertex));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));

    gpuErrchk( hipMemcpy(dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_edge_from, edge_from, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_vertex, g->vertex, size_vertex, hipMemcpyHostToDevice));

    unsigned long long sum = 0;

    //memcpy schedule
    GPUSchedule* dev_schedule;
    gpuErrchk( hipMallocManaged((void**)&dev_schedule, sizeof(GPUSchedule)));
    //dev_schedule->transform_in_exclusion_optimize_group_val(schedule);
    int schedule_size = schedule_iep.get_size();
    int max_prefix_num = schedule_size * (schedule_size - 1) / 2;

    int in_exclusion_optimize_vertex_id_size = schedule_iep.in_exclusion_optimize_vertex_id.size();
    int in_exclusion_optimize_array_size  = schedule_iep.in_exclusion_optimize_coef.size();

    assert(in_exclusion_optimize_array_size == schedule_iep.in_exclusion_optimize_coef.size());
    assert(in_exclusion_optimize_array_size == schedule_iep.in_exclusion_optimize_flag.size());

    printf("array size %d\n", in_exclusion_optimize_array_size);
    fflush(stdout);

    int* in_exclusion_optimize_vertex_id = new int[in_exclusion_optimize_vertex_id_size];
    bool* in_exclusion_optimize_vertex_flag = new bool[in_exclusion_optimize_vertex_id_size];
    int* in_exclusion_optimize_vertex_coef = new int[in_exclusion_optimize_vertex_id_size];

    int* in_exclusion_optimize_coef = new int[in_exclusion_optimize_array_size];
    bool* in_exclusion_optimize_flag = new bool[in_exclusion_optimize_array_size];
    int* in_exclusion_optimize_ans_pos = new int[in_exclusion_optimize_array_size];

    for(int i = 0; i < in_exclusion_optimize_vertex_id_size; ++i) {
        in_exclusion_optimize_vertex_id[i] = schedule_iep.in_exclusion_optimize_vertex_id[i];
        in_exclusion_optimize_vertex_flag[i] = schedule_iep.in_exclusion_optimize_vertex_flag[i];
        in_exclusion_optimize_vertex_coef[i] = schedule_iep.in_exclusion_optimize_vertex_coef[i];
    }

    for(int i = 0; i < in_exclusion_optimize_array_size; ++i) {
        in_exclusion_optimize_coef[i] = schedule_iep.in_exclusion_optimize_coef[i];
        in_exclusion_optimize_flag[i] = schedule_iep.in_exclusion_optimize_flag[i];
        in_exclusion_optimize_ans_pos[i] = schedule_iep.in_exclusion_optimize_ans_pos[i];
    }

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_id, in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_flag, in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_coef, in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_coef, in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_flag, in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size));
    gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_ans_pos, in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->adj_mat, sizeof(int) * schedule_size * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->adj_mat, schedule_iep.get_adj_mat_ptr(), sizeof(int) * schedule_size * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->father_prefix_id, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->father_prefix_id, schedule_iep.get_father_prefix_id_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->last, schedule_iep.get_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->next, schedule_iep.get_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->break_size, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->break_size, schedule_iep.get_break_size_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->loop_set_prefix_id, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->loop_set_prefix_id, schedule_iep.get_loop_set_prefix_id_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_last, schedule_iep.get_restrict_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_next, schedule_iep.get_restrict_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_index, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_index, schedule_iep.get_restrict_index_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    dev_schedule->in_exclusion_optimize_array_size = in_exclusion_optimize_array_size;
    dev_schedule->in_exclusion_optimize_vertex_id_size = in_exclusion_optimize_vertex_id_size;
    dev_schedule->size = schedule_iep.get_size();
    dev_schedule->total_prefix_num = schedule_iep.get_total_prefix_num();
    dev_schedule->basic_prefix_num = schedule_iep.get_basic_prefix_num();
    dev_schedule->total_restrict_num = schedule_iep.get_total_restrict_num();
    dev_schedule->in_exclusion_optimize_num = schedule_iep.get_in_exclusion_optimize_num();
    //dev_schedule->k_val = schedule.get_k_val();

    printf("schedule_iep.prefix_num: %d\n", schedule_iep.get_total_prefix_num());
    printf("shared memory for vertex set per block: %ld bytes\n", 
            (schedule_iep.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int));

    tmpTime.print("Prepare time cost");
    tmpTime.check();

    uint32_t buffer_size = VertexSet::max_intersection_size;
    uint32_t block_shmem_size = (schedule_iep.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);
    //dev_schedule->ans_array_offset = block_shmem_size - in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);
    // 注意：此处没有错误，buffer_size代指每个顶点集所需的int数目，无需再乘sizeof(uint32_t)，但是否考虑对齐？
    //因为目前用了managed开内存，所以第一次运行kernel会有一定额外开销，考虑运行两次，第一次作为warmup

    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm, gpu_pattern_matching, THREADS_PER_BLOCK, block_shmem_size);
    printf("max number of active warps per SM: %d\n", max_active_blocks_per_sm * WARPS_PER_BLOCK);

    gpu_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, block_shmem_size>>>
        (g->e_cnt, buffer_size, dev_edge_from, dev_edge, dev_vertex, dev_tmp);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

    //sum /= schedule.get_in_exclusion_optimize_redundancy();

    printf("count %llu\n", sum);
    tmpTime.print("Counting time cost");
    //之后需要加上hipFree

    // 尝试释放一些内存
    gpuErrchk(hipFree(dev_edge));
    gpuErrchk(hipFree(dev_edge_from));
    gpuErrchk(hipFree(dev_vertex));
    gpuErrchk(hipFree(dev_tmp));

    gpuErrchk(hipFree(dev_schedule->adj_mat));
    gpuErrchk(hipFree(dev_schedule->father_prefix_id));
    gpuErrchk(hipFree(dev_schedule->last));
    gpuErrchk(hipFree(dev_schedule->next));
    gpuErrchk(hipFree(dev_schedule->loop_set_prefix_id));
    gpuErrchk(hipFree(dev_schedule->restrict_last));
    gpuErrchk(hipFree(dev_schedule->restrict_next));
    gpuErrchk(hipFree(dev_schedule->restrict_index));

    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_vertex_id));
    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_coef));
    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_flag));

    gpuErrchk(hipFree(dev_schedule));

    delete[] edge_from;
    delete[] in_exclusion_optimize_vertex_id;
    delete[] in_exclusion_optimize_coef;
    delete[] in_exclusion_optimize_flag;
}

int main(int argc,char *argv[]) {
    Graph *g;
    DataLoader D;

    /*
       if (argc < 2) {
       printf("Usage: %s dataset_name graph_file [binary/text]\n", argv[0]);
       printf("Example: %s Patents ~hzx/data/patents_bin binary\n", argv[0]);
       printf("Example: %s Patents ~zms/patents_input\n", argv[0]);

       printf("\nExperimental usage: %s [graph_file.g]\n", argv[0]);
       printf("Example: %s ~hzx/data/patents.g\n", argv[0]);
       return 0;
       }

       bool binary_input = false;
       if (argc >= 4)
       binary_input = (strcmp(argv[3], "binary") == 0);

       DataType my_type;
       if (argc >= 3) {
       GetDataType(my_type, argv[1]);

       if (my_type == DataType::Invalid) {
       printf("Dataset not found!\n");
       return 0;
       }
       }*/

    using std::chrono::system_clock;
    auto t1 = system_clock::now();

    bool ok;
    /*
       if (argc >= 3) {
    // 注：load_data的第四个参数用于指定是否读取二进制文件输入，默认为false
    ok = D.load_data(g, my_type, argv[2], binary_input);
    } else {
    ok = D.fast_load(g, argv[1]);
    }
    */

    ok = D.fast_load(g, argv[1]);

    if (!ok) {
        printf("data load failure :-(\n");
        return 0;
    }

    auto t2 = system_clock::now();
    auto load_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    printf("Load data success! time: %g seconds\n", load_time.count() / 1.0e6);
    fflush(stdout);

    allTime.check();

    // const char *pattern_str = "0111010011100011100001100"; // 5 house p1
    //const char *pattern_str = "011011101110110101011000110000101000"; // 6 p2
    // const char *pattern_str = "0111111101111111011101110100111100011100001100000"; // 7 p5
    // const char *pattern_str = "0111111101111111011001110100111100011000001100000"; // 7 p6

    int pattern_size = atoi(argv[2]);
    const char* pattern_str= argv[3];

    Pattern p(pattern_size, pattern_str);
    printf("pattern = \n");
    p.print();
    printf("max intersection size %d\n", VertexSet::max_intersection_size);
    bool is_pattern_valid;
    bool use_in_exclusion_optimize = true;
    Schedule_IEP schedule_iep(p, is_pattern_valid, 1, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt);

    if (!is_pattern_valid) {
        printf("pattern is invalid!\n");
        return 0;
    }

    pattern_matching_init(g, schedule_iep);

    allTime.print("Total time cost");

    return 0;
}
