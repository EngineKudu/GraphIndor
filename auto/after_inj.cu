#include "hip/hip_runtime.h"
void pattern_matching_init(Graph *g, const Schedule_IEP& schedule_iep) {
    printf("basic prefix %d, total prefix %d\n", schedule_iep.get_basic_prefix_num(), schedule_iep.get_total_prefix_num());

    int num_blocks = 1024; // TODO: calculate maximum number of blocks dynamically
    int num_total_warps = num_blocks * WARPS_PER_BLOCK;

    size_t size_edge = g->e_cnt * sizeof(uint32_t);
    size_t size_vertex = (g->v_cnt + 1) * sizeof(uint32_t);
    // size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * (schedule_iep.get_total_prefix_num() + 2); //prefix + subtraction + tmp
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * schedule_iep.get_total_prefix_num();

    schedule_iep.print_schedule();
    uint32_t *edge_from = new uint32_t[g->e_cnt];
    for(uint32_t i = 0; i < g->v_cnt; ++i)
        for(uint32_t j = g->vertex[i]; j < g->vertex[i+1]; ++j)
            edge_from[j] = i;

    tmpTime.check(); 

    uint32_t *dev_edge;
    uint32_t *dev_edge_from;
    uint32_t *dev_vertex;
    uint32_t *dev_tmp;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_edge_from, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_vertex, size_vertex));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));

    gpuErrchk( hipMemcpy(dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_edge_from, edge_from, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_vertex, g->vertex, size_vertex, hipMemcpyHostToDevice));

    unsigned long long sum = 0;

    printf("schedule_iep.prefix_num: %d\n", schedule_iep.get_total_prefix_num());

    uint32_t buffer_size = VertexSet::max_intersection_size; // 注意：此处没有错误，buffer_size代指每个顶点集所需的int数目，无需再乘sizeof(uint32_t)，但是否考虑对齐？
    uint32_t block_subtraction_set_size = (schedule_iep.get_size() - schedule_iep.get_in_exclusion_optimize_num()) * WARPS_PER_BLOCK * sizeof(uint32_t);
    uint32_t block_shmem_size = (schedule_iep.get_total_prefix_num() + 1) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + block_subtraction_set_size;
    // printf("block_shmem: %u subtraction reserve: %d offset: %u\n", block_shmem_size, block_subtraction_set_size, dev_schedule->ans_array_offset);
    // ans_array_offset的意义已改变，是block内subtraction_set实际空间的偏移（以字节计）
     
    //因为目前用了managed开内存，所以第一次运行kernel会有一定额外开销，考虑运行两次，第一次作为warmup
    
    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm, gpu_pattern_matching, THREADS_PER_BLOCK, block_shmem_size);
    printf("max number of active warps per SM: %d\n", max_active_blocks_per_sm * WARPS_PER_BLOCK);

    tmpTime.print("Prepare time cost");
    tmpTime.check();
    
    gpu_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, block_shmem_size>>>
        (g->e_cnt, buffer_size, dev_edge_from, dev_edge, dev_vertex, dev_tmp);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

    sum /= schedule_iep.get_in_exclusion_optimize_redundancy();
    
    #ifdef PRINT_ANS_TO_FILE
    freopen("1.out", "w", stdout);
    printf("count %llu\n", sum);
    fclose(stdout);
    #endif
    printf("count %llu\n", sum);
    tmpTime.print("Counting time cost");
    //之后需要加上hipFree

    // 尝试释放一些内存
    gpuErrchk(hipFree(dev_edge));
    gpuErrchk(hipFree(dev_edge_from));
    gpuErrchk(hipFree(dev_vertex));
    gpuErrchk(hipFree(dev_tmp));

    delete[] edge_from;
}

int main(int argc,char *argv[]) {
    Graph *g;
    DataLoader D;

    /*
    if (argc < 2) {
        printf("Usage: %s dataset_name graph_file [binary/text]\n", argv[0]);
        printf("Example: %s Patents ~hzx/data/patents_bin binary\n", argv[0]);
        printf("Example: %s Patents ~zms/patents_input\n", argv[0]);

        printf("\nExperimental usage: %s [graph_file.g]\n", argv[0]);
        printf("Example: %s ~hzx/data/patents.g\n", argv[0]);
        return 0;
    }

    bool binary_input = false;
    if (argc >= 4)
        binary_input = (strcmp(argv[3], "binary") == 0);

    DataType my_type;
    if (argc >= 3) {
        GetDataType(my_type, argv[1]);

        if (my_type == DataType::Invalid) {
            printf("Dataset not found!\n");
            return 0;
        }
    }*/

    using std::chrono::system_clock;
    auto t1 = system_clock::now();

    bool ok;
    /*
    if (argc >= 3) {
        // 注：load_data的第四个参数用于指定是否读取二进制文件输入，默认为false
        ok = D.load_data(g, my_type, argv[2], binary_input);
    } else {
        ok = D.fast_load(g, argv[1]);
    }
    */

    ok = D.fast_load(g, argv[1]);

    if (!ok) {
        printf("data load failure :-(\n");
        return 0;
    }

    auto t2 = system_clock::now();
    auto load_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    printf("Load data success! time: %g seconds\n", load_time.count() / 1.0e6);
    fflush(stdout);

    allTime.check();

    // const char *pattern_str = "0111010011100011100001100"; // 5 house p1
    //const char *pattern_str = "011011101110110101011000110000101000"; // 6 p2
    // const char *pattern_str = "0111111101111111011101110100111100011100001100000"; // 7 p5
    // const char *pattern_str = "0111111101111111011001110100111100011000001100000"; // 7 p6

    int pattern_size = atoi(argv[2]);
    const char* pattern_str= argv[3];

    Pattern p(pattern_size, pattern_str);
    printf("pattern = \n");
    p.print();
    printf("max intersection size %d\n", VertexSet::max_intersection_size);
    bool is_pattern_valid;
    bool use_in_exclusion_optimize = true;
    Schedule_IEP schedule_iep(p, is_pattern_valid, 1, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt);
    Schedule schedule(p, is_pattern_valid, 1, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt); // schedule is only used for getting redundancy
    schedule_iep.set_in_exclusion_optimize_redundancy(schedule.get_in_exclusion_optimize_redundancy());

    if (!is_pattern_valid) {
        printf("pattern is invalid!\n");
        return 0;
    }

    pattern_matching_init(g, schedule_iep);

    allTime.print("Total time cost");

    return 0;
}
